#include "hip/hip_runtime.h"
#include <stdio.h>
// For the CUDA runtime routines (prefixed with "cuda_")
#include <hip/hip_runtime.h>

#ifndef NULL
#define NULL 0
#endif

#define PI 3.14159265358979323846
float TWO_PI = PI + PI;
float HALF_PI = PI / 2.0;
float ONE_AND_HALF_PI = PI + HALF_PI;

#define Tuple float2

int DEBUG = 0; // only for debug on host side
int h_N; // host N
char* grid = NULL; // only used for printing on host side

int d_N; // for use with the algorithm on gpu side


// GPU SIDE CODE ---
void apply_velocity(Tuple & p, Tuple & v, float t) {
  p.x = p.x + v.x * t;
  p.y = p.y + v.y * t;
}
void apply_boundary_collision(Tuple & p, Tuple & v, Tuple & outV) {
  outV.x = (p.x < 0 || p.x > d_N) ? -v.x : v.x;
  outV.y = (p.y < 0 || p.y > d_N) ? -v.y : v.y;
}
float distance(Tuple & p1, Tuple & p2) {
  float dx = p1.x - p2.x;
  float dy = p1.y - p2.y;
  return (float)sqrt(dx*dx + dy*dy);
}

void apply_nbody_collision(Tuple & p1, Tuple & v1, Tuple & p2, Tuple & v2, Tuple & outV) {
  float dx = p2.x - p1.x;
  float dy = p2.y - p1.y;
  float distance = (float)sqrt(dx * dx + dy * dy);
  // Unit vector in the direction of the collision
  float ax = dx / distance;
  float ay = dy / distance;
  // Projection of the velocities in these axes
  float vb1 = (-v1.x * ay + v1.y * ax);
  float va2 = (v2.x * ax + v2.y * ay);
  // New velocity for v1 in these axes (after collision)
  outV.x = va2 * ax - vb1 * ay;
  outV.y = va2 * ay + vb1 * ax;// new vx,vy for particle 1 after collision
}

void update_particle_velocities_boundary_collisions(Tuple positions[], Tuple velocities[]) {
  int i;
  for (i = 0; i < d_N; i++) {
    // writes to second half of velocities array
    apply_boundary_collision(positions[i], velocities[i], velocities[i+d_N]);
  }
}

void update_particle_velocities_nbody_collisions(Tuple positions[], Tuple velocities[]) {
  int i, j;
  int collisions;
  for (i = 0; i < d_N; i++) {
    collisions = 0;
    for (j = 0; j < d_N; j++) {
      if (i != j && distance(positions[i], positions[j]) < 1.0) {
        // reads from second half of array and writes to first half
        apply_nbody_collision(positions[i], velocities[i+d_N], positions[j], velocities[j+d_N],
          velocities[i]);
        collisions++;
      }
    }
    if (collisions == 0) {
      // need to copy over velocity if there was no collision
      velocities[i] = velocities[i+d_N];
    }
  }
}

void update_particle_positions(Tuple positions[], Tuple velocities[], float t) {
  int i;
  for (i = 0; i < d_N; i++) {
    apply_velocity(positions[i], velocities[i], t);
  }
}
// -- END GPU SIDE CODE


// HOST SIDE CODE -----
int grid_index(int i, int j) {
  return (i*(h_N+1))+j;
}

void print(Tuple positions[]) {
  int i, j;
  Tuple p;
  int size = (h_N+2)*(h_N+2); // 1 square buffer on all sides

  if (grid == NULL) {
    grid = (char*)malloc(sizeof(char)*size);
  }
  for (i = 0; i < size; i++) {
    grid[i] = ' ';
  }
  for (i = 0; i < h_N; i++) {
    p = positions[i];
    grid[grid_index((int)p.x+1, (int)p.y)+1] = 'o';
  }
  for (i = 0; i < h_N+2; i++) {
    for (j = 0; j < h_N+2; j++) {
      printf("%c", grid[grid_index(i,j)]);
    }
    printf("\n");
  }
}
void output(Tuple positions[], float t, float step) {
  printf("At time %.2f\n", t);
  print(positions);
}

float rand_float(float range) {
  return range * (float)rand() / (float)RAND_MAX;
}

void simulate(Tuple positions[], Tuple velocities[], float step, int steps) {
  int i;
  float t = 0;
  for (i = 0; i < steps; i++) {
    if (DEBUG > 1) {
      output(positions, t, step);
      t+=step;
    }
    update_particle_velocities_boundary_collisions(positions, velocities);
    update_particle_velocities_nbody_collisions(positions, velocities);
    update_particle_positions(positions, velocities, step);
  }
}

float diff(timespec *start, timespec *stop) {
  float result = (stop->tv_sec - start->tv_sec);
  result += ((float)(stop->tv_nsec - start->tv_nsec)) / (float)1000000000.0;
  return result;
}

void init_random(Tuple positions[], Tuple velocities[]) {
  int i;
  float angle;
  for (i = 0; i < h_N; i++) {
    positions[i].x = (int)rand_float(h_N);
    positions[i].y = i;
    angle = rand_float(TWO_PI);
    velocities[i].x = (float)cos(angle);
    velocities[i].y = (float)sin(angle);
  }
}

int main(int argc, char** args) {
  DEBUG = 2;
  int seed = 0;
  float t = 100.0;
  float step = 0.01;
  int n = 10;
  if (argc > 1) {
    n = atoi(args[1]);
  }
  if (argc > 2) {
    seed = atoi(args[2]);
  }
  if (argc > 3) {
    t = atof(args[3]);
  }
  if (argc > 4) {
    step = atof(args[4]);
  }
  if (argc > 5) {
    DEBUG = atoi(args[5]);
  }

  if (seed > 0) {
    srand(seed);
  }

  int steps = (int)(t / step);

  h_N = n;
  d_N = n;

  Tuple * positions = (Tuple*)malloc(sizeof(Tuple)*h_N);
  Tuple * velocities = (Tuple*)malloc(sizeof(Tuple)*h_N*2);
  init_random(positions, velocities);

  timespec time1, time2;
  clock_gettime(CLOCK_PROCESS_CPUTIME_ID, &time1);

  simulate(positions, velocities, step, steps);
  clock_gettime(CLOCK_PROCESS_CPUTIME_ID, &time2);
  if (DEBUG > 0) {
    int offset = (steps % 2) * h_N;
    output(positions+offset, t, step);
  }
  printf("Total time was %.2f seconds\n", diff(&time1,&time2));
  free(positions);
  free(velocities);
  if (grid != NULL) {
    free(grid);
  }
  return 0;
}
// -- END HOST SIDE CODE
