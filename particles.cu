#include "hip/hip_runtime.h"
#include <stdio.h>
// For the CUDA runtime routines (prefixed with "cuda_")
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#ifndef NULL
#define NULL 0
#endif

#define PI 3.14159265358979323846
float TWO_PI = PI + PI;
float HALF_PI = PI / 2.0;
float ONE_AND_HALF_PI = PI + HALF_PI;

#define Tuple float2

int DEBUG = 0; // only for debug on host side
int h_N; // host N
char* grid = NULL; // only used for printing on host side

__device__ int d_N; // for use with the algorithm on gpu side
Tuple* d_velocities;
Tuple* d_positions;
int THREADS_PER_BLOCK;


// GPU SIDE CODE ---
__device__ void apply_velocity(Tuple & p, Tuple & v, float t) {
  p.x = p.x + v.x * t;
  p.y = p.y + v.y * t;
}
__device__ void apply_boundary_collision(Tuple & p, Tuple & v, Tuple & outV) {
  outV.x = (p.x < 0 || p.x > d_N) ? -v.x : v.x;
  outV.y = (p.y < 0 || p.y > d_N) ? -v.y : v.y;
}
__device__ float distance(Tuple & p1, Tuple & p2) {
  float dx = p1.x - p2.x;
  float dy = p1.y - p2.y;
  return (float)sqrt(dx*dx + dy*dy);
}

__device__ void apply_nbody_collision(Tuple & p1, Tuple & v1, Tuple & p2, Tuple & v2, Tuple & outV) {
  float dx = p2.x - p1.x;
  float dy = p2.y - p1.y;
  float distance = (float)sqrt(dx * dx + dy * dy);
  // Unit vector in the direction of the collision
  float ax = dx / distance;
  float ay = dy / distance;
  // Projection of the velocities in these axes
  float vb1 = (-v1.x * ay + v1.y * ax);
  float va2 = (v2.x * ax + v2.y * ay);
  // New velocity for v1 in these axes (after collision)
  outV.x = va2 * ax - vb1 * ay;
  outV.y = va2 * ay + vb1 * ax;// new vx,vy for particle 1 after collision
}

__global__ void update_particle_velocities_boundary_collisions(Tuple positions[], Tuple velocities[]) {
    int i = threadIdx.x * blockIdx.x;
    apply_boundary_collision(positions[i], velocities[i], velocities[i+d_N]);
}

__global__ void update_particle_velocities_nbody_collisions(Tuple positions[], Tuple velocities[]) {
  int i;
  int j = threadIdx.x * blockIdx.x;
  int collisions;
  for (i = 0; i < d_N; i++) {
    collisions = 0;
      if (i != j && distance(positions[i], positions[j]) < 1.0) {
        // reads from second half of array and writes to first half
        apply_nbody_collision(positions[i], velocities[i+d_N], positions[j], velocities[j+d_N],
          velocities[i]);
        collisions++;
    }
    if (collisions == 0) {
      // need to copy over velocity if there was no collision
      velocities[i] = velocities[i+d_N];
    }
  }
}

__global__ void update_particle_positions(Tuple positions[], Tuple velocities[], float t) {
    int i = threadIdx.x * blockIdx.x;
    apply_velocity(positions[i], velocities[i], t);
}
// -- END GPU SIDE CODE


// HOST SIDE CODE -----
int grid_index(int i, int j) {
  return (i*(h_N+1))+j;
}

void print(Tuple positions[]) {
  int i, j;
  Tuple p;
  int size = (h_N+2)*(h_N+2); // 1 square buffer on all sides

  if (grid == NULL) {
    grid = (char*)malloc(sizeof(char)*size);
  }
  for (i = 0; i < size; i++) {
    grid[i] = ' ';
  }
  for (i = 0; i < h_N; i++) {
    p = positions[i];
    grid[grid_index((int)p.x+1, (int)p.y)+1] = 'o';
  }
  for (i = 0; i < h_N+2; i++) {
    for (j = 0; j < h_N+2; j++) {
      printf("%c", grid[grid_index(i,j)]);
    }
    printf("\n");
  }
}
void output(Tuple positions[], float t, float step) {
  printf("At time %.2f\n", t);
  print(positions);
}

float rand_float(float range) {
  return range * (float)rand() / (float)RAND_MAX;
}

void simulate(Tuple positions[], Tuple velocities[], float step, int steps) {
  hipMalloc((void**)&d_velocities,2*h_N*sizeof(Tuple));
  hipMalloc((void**)&d_positions,h_N*sizeof(Tuple));
  hipMemcpy(d_velocities,velocities,2*h_N*sizeof(Tuple), hipMemcpyHostToDevice);
  hipMemcpy(d_positions,positions,h_N*sizeof(Tuple), hipMemcpyHostToDevice);
  int i;
  float t = 0;
  for (i = 0; i < steps; i++) {
    if (DEBUG > 1) {
      output(positions, t, step);
      t+=step;
    }
    dim3 blocks(h_N/THREADS_PER_BLOCK,1);
    dim3 threads(THREADS_PER_BLOCK,1);
    update_particle_velocities_boundary_collisions<<<blocks,threads>>>(d_positions, d_velocities);
    update_particle_velocities_nbody_collisions<<<blocks,threads>>>(d_positions, d_velocities);
    update_particle_positions<<<blocks,threads>>>(d_positions, d_velocities, step);
  }
  hipMemcpy(velocities,d_velocities,h_N*sizeof(Tuple), hipMemcpyDeviceToHost);
  hipMemcpy(positions,d_positions,h_N*sizeof(Tuple), hipMemcpyDeviceToHost);
}

float diff(timespec *start, timespec *stop) {
  float result = (stop->tv_sec - start->tv_sec);
  result += ((float)(stop->tv_nsec - start->tv_nsec)) / (float)1000000000.0;
  return result;
}

void init_random(Tuple positions[], Tuple velocities[]) {
  int i;
  float angle;
  for (i = 0; i < h_N; i++) {
    positions[i].x = (int)rand_float(h_N);
    positions[i].y = i;
    angle = rand_float(TWO_PI);
    velocities[i].x = (float)cos(angle);
    velocities[i].y = (float)sin(angle);
  }
}

int main(int argc, char** args) {
  DEBUG = 2;
  int seed = 0;
  float t = 100.0;
  float step = 0.01;
  int n = 10;
  if (argc > 1) {
    n = atoi(args[1]);
  }
  if (argc > 2) {
    seed = atoi(args[2]);
  }
  if (argc > 3) {
    t = atof(args[3]);
  }
  if (argc > 4) {
    step = atof(args[4]);
  }
  if (argc > 5) {
    DEBUG = atoi(args[5]);
  }

  if (seed > 0) {
    srand(seed);
  }

  int steps = (int)(t / step);

  h_N = n;
  if (argc > 6) {
	THREADS_PER_BLOCK = atoi(args[6]);
	}
	hipMemcpyToSymbol(HIP_SYMBOL(d_N),&h_N, sizeof(int));

  Tuple * positions = (Tuple*)malloc(sizeof(Tuple)*h_N);
  Tuple * velocities = (Tuple*)malloc(sizeof(Tuple)*h_N*2);
  init_random(positions, velocities);

  timespec time1, time2;
  clock_gettime(CLOCK_PROCESS_CPUTIME_ID, &time1);

  simulate(positions, velocities, step, steps);
  clock_gettime(CLOCK_PROCESS_CPUTIME_ID, &time2);
  if (DEBUG > 0) {
    int offset = (steps % 2) * h_N;
    output(positions+offset, t, step);
  }
  printf("Total time was %.2f seconds\n", diff(&time1,&time2));
  free(positions);
  free(velocities);
  if (grid != NULL) {
    free(grid);
  }
  return 0;
}
// -- END HOST SIDE CODE
